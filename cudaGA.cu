/**
 * description:
 * 	Simple genetic algorithm for finding equation that equals target number
 * 	example for number 27 has many results:
 * 		- 7+29-8-1
 * 		- 15+28-16
 * 	this code isn't really perfect ^_^ but shows some basics.
 * 	tested in linux fedora 14 with valgrind.
 * 
 * author: ADRABI Abderrahim (adrabi[at]mail[dot]ru)
 * date: 2011-10-03
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#define POPSIZE		1024
#define ELITRATE	0.1f
#define MUTATIONRATE	0.25f
#define ELEMENTS	8
#define MUTATION	RAND_MAX * MUTATIONRATE
#define TARGET		5270



 #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * Basic elements for construction an equation
 */
static const char *BIN_ELEMENTS[12] = {
  "0000\0",			// 0
  "0001\0",			// 1
  "0010\0",			// 2
  "0011\0",			// 3
  "0100\0",			// 4 
  "0101\0",			// 5 
  "0110\0",			// 6
  "0111\0",			// 7
  "1000\0",			// 8
  "1001\0",			// 9
  "1010\0",			// + 
  "1011\0"			// -
};

/**
 * Structure base of genome
 */
typedef struct
{
  unsigned int fitness;
  char *gen;
} ga_struct;

char* dev_pop;
ga_struct* dev_betapop;
char *dev_gen;

__device__ int
stringCompare (char str1[], char str2[])
{
	int c = 0;
	while(str1[c] == str2[c])
	  {
	  	if(str1[c]=='\0' && str2[c]=='\0'){return 1;}
	  	//printf("equals");
	  	c++;
	  }
	  return 0;
}

/**
 * Initialize new random population
 */
void
init_population (ga_struct * population, ga_struct * beta_population)
{
  const int bin_size = (sizeof (char) * ELEMENTS * 4) + 1;
  int index = 0;
  for (; index < POPSIZE; index++)
    {
      // default initialization/ create empty genome
      population[index].fitness = 0;
      population[index].gen = (char*)malloc (bin_size);
      *population[index].gen = '\0';

      // default initialization/ create empty genome
      beta_population[index].fitness = 0;
      beta_population[index].gen = (char*)malloc (bin_size);
      *beta_population[index].gen = '\0';

      int e = 0;
      for (; e < ELEMENTS; e++)
	{
	  // put random element in population
	  // 12 is count of elements in BIN_ELEMENTS array
	  strcat (population[index].gen, BIN_ELEMENTS[(rand () % 12)]);
	}
    }
}



__global__ void
cal_fitness (char population[])
{
	//printf("inside function");
	//printf("population = %s", population[0]);
	int e = 8;
	int p = 1024;
	int t = 5270;
  
  int index = 0;
  int unsigned fitness = 0;
  for (; index < p; index++)
    {
    	//printf("inside for loop");
      char *gen_str = population;

      //printf("igen, %s", population);
      int sum = 0, current_value = 0, step = 0;
      unsigned int last_operator_index = -1;
      char last_operator = (char) 0;

      for (; step < e; step++)
	{
		//printf("inside 2nd for loop");
	  
	  //element[4] = "\0";
	  //strncpy (element, gen_str, 4);
	  //printf("gen1, %c", gen_str[0]);
	  //printf("gen2, %c", gen_str[1]);
	  //printf("gen3, %c", gen_str[2]);
	  //printf("gen4, %c", gen_str[3]);
	  char element[4];
	  element[0] = gen_str[0];
	  element[1] = gen_str[1];
	  element[2] = gen_str[2];
	  element[3] = gen_str[3];
	  //printf("element, %c", element[0]);
	  //printf("element, %c", element[1]);
	  //printf("element, %c", element[2]);
	  //printf("element, %c", element[3]);
	  //element[4] = '\0';
	  
	  char test[] = "0000\0";
	  char test1[] = "0001\0";
	  char test2[] = "0010\0";
	  char test3[] = "0011\0";
	  char test4[] = "0100\0";
	  char test5[] = "0101\0";
	  char test6[] = "0110\0";
	  char test7[] = "0111\0";
	  char test8[] = "1000\0";
	  char test9[] = "1001\0";
	  char test10[] = "1010\0";
	  char test11[] = "1011\0";
	  //printf("made it this far");
	  if(stringCompare(element,test))
	  //if (strcmp ("0000", element) == 0)
	    {
	      current_value *= 10;
	    }
	    else if (stringCompare(element,test1))
	    {
	      current_value = (current_value * 10) + 1;
	    }
	  else if (stringCompare(element,test2))
	    {
	      current_value = (current_value * 10) + 2;
	    }
	  else if (stringCompare(element,test3))
	    {
	      current_value = (current_value * 10) + 3;
	    }
	  else if (stringCompare(element,test4))
	    {
	      current_value = (current_value * 10) + 4;
	    }
	  else if (stringCompare(element,test5))
	    {
	      current_value = (current_value * 10) + 5;
	    }
	  else if (stringCompare(element,test6))
	    {
	      current_value = (current_value * 10) + 6;
	    }
	  else if (stringCompare(element,test7))
	    {
	      current_value = (current_value * 10) + 7;
	    }
	  else if (stringCompare(element,test8))
	    {
	      current_value = (current_value * 10) + 8;
	    }
	  else if (stringCompare(element,test9))
	    {
	      current_value = (current_value * 10) + 9;
	    }
	  
	  else if (stringCompare(element,test10)
		   && step - last_operator_index > 1 && step + 1 < e)
	    {
	      if (last_operator == (char) 0)
		{
		  sum = current_value;
		}
	      else if (last_operator == '+')
		{
		  sum += current_value;
		}
	      else if (last_operator == '-')
		{
		  sum -= current_value;
		}

	      current_value = 0;
	      last_operator_index = step;
	      last_operator = '+';
	    }
	  
	  else if (stringCompare(element,test11)
		   && step - last_operator_index > 1 && step + 1 < e)
	    {
	      if (last_operator == (char) 0)
		{
		  sum = current_value;
		}
	      else if (last_operator == '+')
		{
		  sum += current_value;
		}
	      else if (last_operator == '-')
		{
		  sum -= current_value;
		}

	      current_value = 0;
	      last_operator_index = step;
	      last_operator = '-';
	    }
	  else
	    {
	      /// error the binary string not found
	    	//printf("whoops");
	      sum = 999999;
	      break;
	    }
	  gen_str += 4;
	}
	//printf("outside for loop");

      if (last_operator == '+')
	{
	  sum += current_value;
	}
      else if (last_operator == '-')
	{
	  sum -= current_value;
	}

      // abs, because fitness is unsigned integer ^_^
      //population[index].fitness = abs (sum - t);           //fix this line
      printf ("fitness = %d", abs (sum - t));
    }
}

/**
 * sort function needed by quick sort
 */
int
sort_func (const void *e1, const void *e2)
{
  return ((ga_struct *) e1)->fitness - ((ga_struct *) e2)->fitness;
}

/**
 * sort population by fitness
 */
inline void
sort_by_fitness (ga_struct * population)
{
  qsort (population, POPSIZE, sizeof (ga_struct), sort_func);
}

/**
 * select elit element in top array after sort
 */
void
elitism (ga_struct * population, ga_struct * beta_population, int esize)
{
  const int gen_len = ELEMENTS * 4 + 1;
  int index = 0;
  for (; index < esize; index++)
    {
      int e = 0;
      for (; e < gen_len; e++)
	{
	  beta_population[index].gen[e] = population[index].gen[e];
	}
    }
}

/**
 * mutate an individual with random rate
 */
void
mutate (ga_struct * member)
{
  int tsize = strlen (member->gen);
  int number_of_mutations = rand () % 10;
  int m = 0;
  for (; m < number_of_mutations; m++)
    {
      int apos = rand () % tsize;

      if (member->gen[apos] == '0')
	{
	  member->gen[apos] = '1';
	}
      else
	{
	  member->gen[apos] = '0';
	}
    }
}

/**
 * mate randomly the rest of population after elitism
 */
void
mate (ga_struct * population, ga_struct * beta_population)
{
  int esize = POPSIZE * ELITRATE;

  // elitism of top elements in array
  elitism (population, beta_population, esize);

  // mate the rest of shitty population xD
  int m = esize, i1 = -1, i2 = -1, pos = -1, tsize = ELEMENTS * 4 + 1;
  for (; m < POPSIZE; m++)
    {
      pos = rand () % tsize;
      i1 = rand () % POPSIZE;
      i2 = rand () % POPSIZE;

      int i = 0;
      for (; i < pos; i++)
	{
	  beta_population[m].gen[i] = population[i1].gen[i];
	}
      for (i = pos; i < tsize; i++)
	{
	  beta_population[m].gen[i] = population[i2].gen[i];
	}

      if (rand () < MUTATION)
	{
	  mutate (&beta_population[m]);
	}
    }
}

/**
 * decode binary string to readable format
 */
void
decode_gen (ga_struct * member)
{
  char *gen_str = member->gen;
  int step = 0;

  for (; step < ELEMENTS; step++)
    {
      char element[5] = "\0";
      strncpy (element, gen_str, 4);
      if (strcmp ("0000", element) == 0)
	{
	  printf ("0");
	}
      else if (strcmp ("0001", element) == 0)
	{
	  printf ("1");
	}
      else if (strcmp ("0010", element) == 0)
	{
	  printf ("2");
	}
      else if (strcmp ("0011", element) == 0)
	{
	  printf ("3");
	}
      else if (strcmp ("0100", element) == 0)
	{
	  printf ("4");
	}
      else if (strcmp ("0101", element) == 0)
	{
	  printf ("5");
	}
      else if (strcmp ("0110", element) == 0)
	{
	  printf ("6");
	}
      else if (strcmp ("0111", element) == 0)
	{
	  printf ("7");
	}
      else if (strcmp ("1000", element) == 0)
	{
	  printf ("8");
	}
      else if (strcmp ("1001", element) == 0)
	{
	  printf ("9");
	}
      else if (strcmp ("1010", element) == 0)
	{
	  printf ("+");
	}
      else if (strcmp ("1011", element) == 0)
	{
	  printf ("-");
	}
      gen_str += 4;
    }

  printf ("\n");
}

/**
 * free memory before exit program
 */
__global__ void
free_population (ga_struct * population)
{
  int index = 0;
  for (; index < POPSIZE; index++)
    {
      free (population[index].gen);
    }
  free (population);
}

/**
 * swap arrays pointers
 */
void
swap (ga_struct ** p1, ga_struct ** p2)
{
  ga_struct *tmp = *p1;
  *p1 = *p2;
  *p2 = tmp;
}



/**
 * main program
 */
int
main (void)
{
	//printf("begin");
	float cpu1,cpu2;	
	cpu1 = ((float) clock())/CLOCKS_PER_SEC;
  srand (time (NULL));

  ga_struct *population = (ga_struct*)malloc (sizeof (ga_struct) * POPSIZE);
  ga_struct *beta_population = (ga_struct*)malloc (sizeof (ga_struct) * POPSIZE);


  init_population (population, beta_population);
 
  //cudaMalloc((ga_struct**)&dev_betapop,sizeof (ga_struct) * POPSIZE);
  dim3 numBlocks(1,1);
  dim3 threads_per_block(1,1);


  //printf("test gen2, %s", population[0].gen);
dev_gen = (char*)malloc(sizeof(char)*POPSIZE);
char *dev_gen[POPSIZE];
int i = 0;
  for (; i < POPSIZE; i++)
    {
    	dev_gen[i] = population[i].gen;
	}

	//printf("test gen, %s", dev_gen[0]);
	//printf("test gen2, %s", population[i].gen);
	

  int index = 0;
  for (; index < POPSIZE; index++)
    {
      //printf("before func");
      //cudaMemcpy(dev_pop,population,sizeof(ga_struct)*POPSIZE,cudaMemcpyHostToDevice);
      //cudaMemcpy(dev_gen,(*population).gen,sizeof(char)*POPSIZE,cudaMemcpyHostToDevice);
      //cudaMemcpy(&((*dev_pop).gen),&dev_gen,sizeof(char)*POPSIZE,cudaMemcpyHostToDevice);
      
      
       
  		hipMalloc((void**)&dev_pop,sizeof (char) * POPSIZE);
  		//cudaMalloc(&dev_gen,sizeof (char) * POPSIZE);

      

      hipMemcpy(dev_pop,dev_gen,sizeof(char)*POPSIZE,hipMemcpyHostToDevice);
      
      cal_fitness<<<1,1>>>(dev_pop);
      //printf("test dev pop = %s", dev_gen[0]);

      //gpuErrchk( cudaPeekAtLastError() );
      //printf("after func");
      //cal_fitness((ga_struct*)population);
      hipMemcpy(dev_gen,dev_pop,sizeof(char)*POPSIZE,hipMemcpyDeviceToHost);
      sort_by_fitness (population);
      

      // print current best individual
      printf ("binary string: %s - fitness: %d\n", population[0].gen,
	      population[0].fitness);

      if (population[0].fitness == 0)
	{
	  //~ print equation
	  decode_gen (&population[0]);
	  break;
	}
	  
      mate (population, beta_population);
      swap (&population, &beta_population);
      
    }
  
  free_population<<<numBlocks,threads_per_block>>>((ga_struct*)population);
  free_population<<<numBlocks,threads_per_block>>>((ga_struct*)beta_population);
cpu2 = ((float) clock())/CLOCKS_PER_SEC;
  

  printf("Execution time (s) = %le\n",cpu2-cpu1);

  return 0;
}

